#include "hip/hip_runtime.h"
#include <stdio.h>
#include <opencv2/opencv.hpp>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <inttypes.h>


#define MAX_DISP 1000
#define NCHANS 3
#define IDP_LVL 4

#define BLOCK_SIZE 16


// timing utility
struct timespec check_timer(const char* str, struct timespec* ts){
	struct timespec oldtime;
	// copy old time over
	oldtime.tv_nsec = ts->tv_nsec;
	oldtime.tv_sec = ts->tv_sec;
	// update ts
	clock_gettime(CLOCK_REALTIME, ts);
	// print old time
	int diffsec;
	int diffnsec;
	if(str != NULL){
		diffsec =  ts->tv_sec - oldtime.tv_sec;
		diffnsec =  ts->tv_nsec - oldtime.tv_nsec;
		// correct the values if we measured over an integer second break:
		if(diffnsec < 0){
			diffsec--;
			diffnsec += 1000000000;
		}
		printf("%s:%ds %dns\n",str,diffsec,diffnsec);
	}
	return (struct timespec) {diffsec, diffnsec};
}

// little bitty kernel to initialize blocks of device memory
__global__ void gpu_memset(unsigned char* start, unsigned char value, int length){
	int tx = threadIdx.x;
	int bx = blockIdx.x;
	int gx = bx*blockDim.x + tx;
	if(gx < length){
		start[gx] = value;
	}
}

// teeny little helper function
void gpu_perror(char* input){
	printf("%s: %s\n", input, hipGetErrorString(hipGetLastError()));


// In the future it may be useful to bring a whole line of pixels into local memory...
// ... from shared memory, and then do everything that needs to be dones with that line...
// ... for a given pixel, before moving to the next row...
// ... or maybe it would be better to use a single location of spacial sigma.  Oh I like that. 


// now let's try just running with 32 threads, but each 32 thread warp stretches horizontally across a row
__global__ void asw_kernel2(unsigned char* global_left, unsigned char* global_right, unsigned char* output, unsigned char* debug,
	int nrows, int ncols, int ndisp, int win_size, int win_rad, float s_sigma, float c_sigma)
{
	// now that we are just doing a line at a time, we don't really need dynamic allocations
	__shared__ unsigned char ref[(win_size + blockDim.x)*NCHANS*IDP_LVL];
	__shared__ unsigned char tgt[(win_size + blockDim.x + MAX_DISP)*NCHANS*IDP_LVL];
	// if we start somewhere with a middle row of the image, then we can use a shared variable to share center values
	__shared__ unsigned char ref_center_pix[NCHANS*IDP_LVL];
	__shared__ unsigned char tgt_center_pix[MAX_DISP*NCHANS*IDP_LVL]

	int ref_width_bytes = (2*win_rad+blockDim.x)*NCHANS*sizeof(unsigned char);
	int tgt_width_bytes = (ndisp+2*win_rad+blockDim.x)*NCHANS*sizeof(unsigned char);

	// we are sticking with local memory for the sums of the disparities, because since we only access that occasionally I don't think the latency is a problem
	float costs[MAX_DISP*IDP_LVL];
	float weights[MAX_DISP*IDP_LVL];

	// other things should fall into register memory


}

// Device code
__global__ void asw_kernel(unsigned char* global_left, unsigned char* global_right, unsigned char* output, unsigned char* debug,
	int nrows, int ncols, int ndisp, int win_size, int win_rad, float s_sigma, float c_sigma)
	{
	// ok, we're going to try a block size of 32 ( 32x32 = 1024, max threads per block )
	// no... we'll use 16x16 since there's problems with shared memory with two images
	// each thread will calculate the full asw stereo output for a single pixel
	// shared memory will contain all the input image data for the full block of asw calculations
	// texture memory will contain the spacial filter, eventually
	extern __shared__ unsigned char ref[]; // contains both left and right image data

	// get the size of the sub-images that we are considering
	// reference window
	int ref_width_bytes = (2*win_rad+blockDim.x)*NCHANS*sizeof(unsigned char);
	// int ref_rows = (2*win_rad+blockDim.y);
	// target window
	int tgt_width_bytes = (ndisp+2*win_rad+blockDim.x)*NCHANS*sizeof(unsigned char);
	// int tgt_rows = (2*win_rad+blockDim.y);

	unsigned char* tgt = (unsigned char*)(&ref[ ref_width_bytes*(2*win_rad+blockDim.y) ]); // tgt image, reference to somwhere of shared allocated memory

	float ref_c_factor;
	float tgt_c_factor;
	float s_factor;
	float ref_c2p_diff;
	float tgt_c2p_diff;
	float ref2tgt_diff;
	// variables for keeping track of the output
	float weight;
	float cost;
	float min_cost;
	unsigned char min_cost_index;
	unsigned char ref_center_pix[3];
	unsigned char tgt_center_pix[3];
	unsigned char ref_pix[3];
	unsigned char tgt_pix[3];

	int disp;
	int win_x;
	int win_y;
	int dx;
	int tgt_x;

	// get identity of this thread (changing these to #define's)

	#define tx (threadIdx.x)
	#define ty (threadIdx.y)
	#define bx (blockIdx.x + 5)
	#define by (blockIdx.y + 1)
	#define gx (bx*blockDim.x + tx)
	#define gy (by*blockDim.y + ty)

	// setup LUTs // nevermind... right now there are none

	// copy relevant subimages to shared memory
	// TODO: additional boundary checks on this data
	// TODO: better division technique
	// TODO: investigate where syncthreads() needs to be called for best performance
	// we can copy the 24-bit image over 32 bits at a time
	// except then I don't know how to deal with the edge case
	// so let's just do one character at a time
	// starting with reference image: (4 deleted register variables)	
	// int xblocks = (ref_width_bytes / blockDim.x + 1);
	// int yblocks = ((2*win_rad+blockDim.y) / blockDim.y + 1);
	// int xstart = ((bx*blockDim.x - win_rad)*NCHANS);
	// int ystart = (gy - win_rad);
	// 29 variables here
	for(win_x = 0; win_x < (ref_width_bytes / blockDim.x + 1); win_x++){
		// int x_idx = (win_x*blockDim.x + tx);
		// int g_x_idx = (((bx*blockDim.x - win_rad)*NCHANS) + win_x*blockDim.x + tx);
		if((win_x*blockDim.x + tx) < ref_width_bytes){
			for(win_y = 0; win_y < ((2*win_rad+blockDim.y) / blockDim.y + 1); win_y++){
				// int y_idx = (win_y*blockDim.y + ty);
				// int g_y_idx = ((gy - win_rad) + win_y*blockDim.y);
				if((win_y*blockDim.y + ty) < (2*win_rad+blockDim.y)){
					// copy bytes (not pixels) from global_left into reference image
					ref[(win_y*blockDim.y + ty)*ref_width_bytes + (win_x*blockDim.x + tx)] = global_left[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad)*NCHANS) + win_x*blockDim.x + tx)];
					// copy into the debug image (only made to work with a single block of threads)
					// debug[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad)*NCHANS) + win_x*blockDim.x + tx)]  = ref[(win_y*blockDim.y + ty)*ref_width_bytes + (win_x*blockDim.x + tx)];
				}
			}
		}
	}
	// then to the target image: (4 deleted register variables)
	// xblocks = (tgt_width_bytes / blockDim.x + 1);
	// yblocks = ((2*win_rad+blockDim.y) / blockDim.y + 1);
	// xstart = ((bx*blockDim.x - win_rad - ndisp)*NCHANS);
	// ystart = (gy - win_rad);
	for(win_x = 0; win_x < (tgt_width_bytes / blockDim.x + 1); win_x++){
		// int x_idx = (win_x*blockDim.x + tx);
		// int g_x_idx = (((bx*blockDim.x - win_rad - ndisp)*NCHANS) + win_x*blockDim.x + tx);
		if((win_x*blockDim.x + tx) < tgt_width_bytes){
			for(win_y = 0; win_y < ((2*win_rad+blockDim.y) / blockDim.y + 1); win_y++){
				// int y_idx = (win_y*blockDim.y + ty);
				// int g_y_idx = ((gy - win_rad) + win_y*blockDim.y);
				if((win_y*blockDim.y + ty) < (2*win_rad+blockDim.y)){
					// copy bytes (not pixels) from global_left into reference image
					tgt[(win_y*blockDim.y + ty)*tgt_width_bytes + (win_x*blockDim.x + tx)] = global_right[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad - ndisp)*NCHANS) + win_x*blockDim.x + tx)];
					// copy into the debug image (only made to work with a single block of threads)
					// debug[((gy - win_rad) + win_y*blockDim.y)*ncols*NCHANS + (((bx*blockDim.x - win_rad - ndisp)*NCHANS) + win_x*blockDim.x + tx)]  = tgt[(win_y*blockDim.y + ty)*tgt_width_bytes + (win_x*blockDim.x + tx)];
				}
			}
		}
	}

	__syncthreads();

	// get a pointer to the ref_center_pix, which is constant for any given thread
	ref_center_pix[0] = ref[(win_rad + ty)*ref_width_bytes + (win_rad + tx)*NCHANS + 0];
	ref_center_pix[1] = ref[(win_rad + ty)*ref_width_bytes + (win_rad + tx)*NCHANS + 1];
	ref_center_pix[2] = ref[(win_rad + ty)*ref_width_bytes + (win_rad + tx)*NCHANS + 2];
	// initialize min_cost to some arbitrarily large value
	min_cost = 1e12;
	// initialize min_cost_index to 0
	min_cost_index = 0;

	// for each value of ndisp	
	for(disp = 0; disp < ndisp; disp++){
		// get a pointer to the tgt_center_pix, which is constant for each disp
		tgt_center_pix[0] = tgt[(win_rad + ty)*tgt_width_bytes + (ndisp + win_rad + tx - disp)*NCHANS + 0];
		tgt_center_pix[1] = tgt[(win_rad + ty)*tgt_width_bytes + (ndisp + win_rad + tx - disp)*NCHANS + 1];
		tgt_center_pix[2] = tgt[(win_rad + ty)*tgt_width_bytes + (ndisp + win_rad + tx - disp)*NCHANS + 2];
		// reset weight and cost
		weight = 0;
		cost = 0;
		// in each row in the window:
		for(win_x = 0; win_x < win_size; win_x++){
			// locate the pixel in the ref image (deleted this var)
			dx = win_x + tx;
			// locate the pixel in the tgt image (deleted this var)
			tgt_x = ndisp + win_x + tx - disp;
			// find the window-center to pixel x-distance (deleted this var)
			// int dx = win_x - win_rad;
			// in each column of the window:
			for(win_y = 0; win_y < win_size; win_y++){
				// locate the pixel in the ref image (deleted this var)
				// int ref_y = win_y + ty;
				// find the window-center to pixel y-distance (deleted this var)
				// int dy = win_y - win_rad;
				// get the radius^2 value (deleted this var)
				// float radius_2 = (win_x-win_rad)*(win_x-win_rad) + (win_y-win_rad)*(win_y-win_rad);
				// get the s_factor for this particular window location
				s_factor = __expf(-((win_x-win_rad)*(win_x-win_rad) + (win_y-win_rad)*(win_y-win_rad))/(2.*s_sigma*s_sigma));
				// store tgt and ref pixels in register memory
				ref_pix[0] = ref[(win_y+ty)*ref_width_bytes + (dx)*NCHANS + 0];
				ref_pix[1] = ref[(win_y+ty)*ref_width_bytes + (dx)*NCHANS + 1];
				ref_pix[2] = ref[(win_y+ty)*ref_width_bytes + (dx)*NCHANS + 2];
				tgt_pix[0] = tgt[(win_y+ty)*tgt_width_bytes + (tgt_x)*NCHANS + 0];
				tgt_pix[1] = tgt[(win_y+ty)*tgt_width_bytes + (tgt_x)*NCHANS + 1];
				tgt_pix[2] = tgt[(win_y+ty)*tgt_width_bytes + (tgt_x)*NCHANS + 2];
				// get the center-to-pixel and overall color differences (organized together for IDP)
				ref_c2p_diff = abs(ref_center_pix[0] - ref_pix[0]);
				tgt_c2p_diff = abs(tgt_center_pix[0] - ref_pix[0]);
				ref2tgt_diff = abs(ref_pix[0] - tgt_pix[0]);
				ref_c2p_diff += abs(ref_center_pix[1] - ref_pix[1]);
				tgt_c2p_diff += abs(tgt_center_pix[1] - ref_pix[1]);
				ref2tgt_diff+= abs(ref_pix[1] - tgt_pix[1]);
				ref_c2p_diff += abs(ref_center_pix[2] - ref_pix[2]);
				tgt_c2p_diff += abs(tgt_center_pix[2] - ref_pix[2]);
				ref2tgt_diff+= abs(ref_pix[2] - tgt_pix[2]);
				// get the c_factors
				ref_c_factor = __expf(-ref_c2p_diff*ref_c2p_diff/(2.*c_sigma*c_sigma));
				tgt_c_factor = __expf(-tgt_c2p_diff*tgt_c2p_diff/(2.*c_sigma*c_sigma));
				// calulate the pix_weight (this variable has been done away with to increase ILP)
				// pix_weight = s_factor*ref_c_factor*tgt_c_factor;
				// add in the cost
				cost += s_factor*ref_c_factor*tgt_c_factor*ref2tgt_diff;
				// add in the weight
				weight += s_factor*ref_c_factor*tgt_c_factor;
			}
		}
		// now that the window is done, compare this cost (after normalizing) to min_cost
		if( min_cost > cost / weight){
			min_cost = cost / weight;
			min_cost_index = disp;
		}
		__syncthreads();
	}

	// set the output to the index of min_cost
	output[gy*ncols + gx] = min_cost_index;
}

int asw(cv::Mat im_l, cv::Mat im_r, int ndisp, int s_sigma, int c_sigma){
	// window size and win_rad
	int win_size = 3*s_sigma;
	int win_rad = (win_size - 1)/2;
	// declare timer
	struct timespec timer;

	// check that images are matching dimensions
	if(im_l.rows != im_r.rows){
		printf("Error: im_l and im_r do not have matching row count\n");
		return 1;
	}
	if(im_l.cols != im_r.cols){
		printf("Error: im_l and im_r do not have matching col count\n");
		return 1;
	}
	if(im_l.channels() != im_r.channels()){
		printf("Error: im_l and im_r do not have matching channel count\n");
		return 1;
	}

	// set easy-access variables for number of rows, cols, and chans
	int nrows = im_l.rows;
	int ncols = im_l.cols;
	int nchans = im_l.channels();
	// initialize the device input arrays
	unsigned char* d_im_l;
	hipMalloc(&d_im_l,nchans*nrows*ncols*sizeof(unsigned char));
	unsigned char* d_im_r;
	hipMalloc(&d_im_r,nchans*nrows*ncols*sizeof(unsigned char));
	// initialize the output data matrix
	unsigned char* out = (unsigned char*)malloc(nrows*ncols*sizeof(unsigned char));
	unsigned char* d_out;
	hipMalloc(&d_out,nrows*ncols*sizeof(unsigned char));
	unsigned char* debug = (unsigned char*)malloc(nrows*ncols*nchans*sizeof(unsigned char));
	unsigned char* d_debug;
	hipMalloc(&d_debug,nchans*nrows*ncols*sizeof(unsigned char));

	// define a shortcut to the host data arrays
	unsigned char* data_l = ((unsigned char*)(im_l.data));
	unsigned char* data_r = ((unsigned char*)(im_r.data));

	//copy the host input data to the device
    hipMemcpy(d_im_l, data_l, nchans*nrows*ncols*sizeof(unsigned char), hipMemcpyHostToDevice);
    hipMemcpy(d_im_r, data_r, nchans*nrows*ncols*sizeof(unsigned char), hipMemcpyHostToDevice);

	// get gaussian kernel for spacial look-up table:
	// equation from cv::getGaussianKernel(), but without normalization
	// float s_weights[win_size][win_size]; 
	// for(int i=0; i<win_size; i++){
	// 	for(int j=0; j<win_size; j++){
	// 		float x = i-win_rad;
	// 		float y = j-win_rad;
	// 		float radius = sqrt(x*x+y*y);
	// 		s_weights[i][j] = std::pow(2.71828,-radius*radius/(2.*s_sigma*s_sigma));
	// 		// printf("%.6f ",s_weights[i][j]);
	// 	}
	// 	// printf("\n");
	// }

	// get gaussian kernel for color look-up table:
	// equation from cv::getGaussianKernel(), but without normalization
	// float c_weights[511]; 
	// for(int i=0; i<511; i++){
	// 	float radius = i-255;
	// 	c_weights[i] = std::pow(2.71828,-radius*radius/(2.*c_sigma*c_sigma));
	// 	// printf("%.6f ",c_weights[i]);
	// }

	// initialize the outputs (otherwise changes persist between runtimes, hard to debug):
	int tpb = 1024;
	int bpg = nrows*ncols*sizeof(unsigned char) / tpb + 1;
	printf("zeroing output images\n");
	gpu_memset<<<bpg, tpb>>>(d_out,25,nrows*ncols*sizeof(unsigned char));
	gpu_perror("memset1");
	gpu_memset<<<nchans*bpg, tpb>>>(d_debug,25,nchans*nrows*ncols*sizeof(unsigned char));
	gpu_perror("memset2");

	// check some values before calling the asw_kernel
	size_t reference_window_size = (2*win_rad+BLOCK_SIZE)*(2*win_rad+BLOCK_SIZE)*sizeof(unsigned char)*nchans;
	size_t target_window_size = (2*win_rad+ndisp+BLOCK_SIZE)*(BLOCK_SIZE+2*win_rad)*sizeof(unsigned char)*nchans;
	size_t shared_size = target_window_size+reference_window_size;
	printf("win_size %d win_rad %d ndisp %d shared size = %d\n",win_size,win_rad,ndisp,shared_size);
	if(shared_size > 47000){
		printf("FATAL ERROR: shared_size for asw_kernel exceeds the device limit (48 kB), exiting\n");
		return 1;
	}

	// call the asw_kernel
	dim3 blocksPerGrid(22,21);
	dim3 threadsPerBlock(BLOCK_SIZE,BLOCK_SIZE);
	// __global__ void asw_kernel(unsigned char* global_left, unsigned char* global_right, unsigned char* output, unsigned char* debug,
	//		int nrows, int ncols, int nchans, int ndisp, int win_size, int win_rad, float s_sigma, float c_sigma)
	printf("starting asw kernel\n");
	check_timer(NULL,&timer);
    asw_kernel<<<blocksPerGrid, threadsPerBlock, shared_size>>>(d_im_l, d_im_r, d_out, d_debug,
    	nrows, ncols, nchans, ndisp, win_size, win_rad, s_sigma, c_sigma);
    hipDeviceSynchronize();
    check_timer("asw kernel finished",&timer);
	gpu_perror("asw_kernel");

	// copy the device output data to the host
	check_timer(NULL,&timer);
    hipMemcpy(out, d_out, nrows*ncols*sizeof(unsigned char), hipMemcpyDeviceToHost);
    hipMemcpy(debug, d_debug, nrows*ncols*nchans*sizeof(unsigned char), hipMemcpyDeviceToHost);
    check_timer("copying complete",&timer);

    // make an image and view it:
    cv::Mat im_out(nrows,ncols,CV_8UC1,out);
    cv::Mat im_debug(nrows,ncols,CV_8UC3,debug);
    // cv::rectangle(im_debug,cv::Point(16*15,16*15),cv::Point(16*16,16*16),cv::Scalar(255,0,0));
    // cv::rectangle(im_out,cv::Point(16*15,16*15),cv::Point(16*16,16*16),127);
    // cv::imshow("window",im_debug);
    // cv::waitKey(0);
    cv::imshow("window",im_out);
    cv::waitKey(0);

	// cleanup memory
	hipFree(d_im_l);
	hipFree(d_im_r);
	hipFree(d_out);
	hipFree(d_debug);
	free(out);
	free(debug);

	return 0;
}

int main(int argc, char** argv){
	// spacial and color sigmas
	int s_sigma, c_sigma;
	// number of disparities to check
	int ndisp;
	// input images
	cv::Mat im_l, im_r;

	if(argc < 6){
		printf("usage: %s <left image> <right image> <num disparities> <spacial sigma> <color sigma>",argv[0]);
		return 1;
	}else{
		im_l = cv::imread(argv[1]);
		im_r = cv::imread(argv[2]);
		ndisp = atoi(argv[3]);
		s_sigma = atoi(argv[4]);
		c_sigma = atoi(argv[5]);
	}

	return asw(im_l, im_r, ndisp, s_sigma, c_sigma);
}