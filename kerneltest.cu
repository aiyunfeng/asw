
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

struct timespec check_timer(const char* str, struct timespec* ts){
	struct timespec oldtime;
	// copy old time over
	oldtime.tv_nsec = ts->tv_nsec;
	oldtime.tv_sec = ts->tv_sec;
	// update ts
	clock_gettime(CLOCK_REALTIME, ts);
	// print old time
	int diffsec;
	int diffnsec;
	if(str != NULL){
		diffsec =  ts->tv_sec - oldtime.tv_sec;
		diffnsec =  ts->tv_nsec - oldtime.tv_nsec;
		// correct the values if we measured over an integer second break:
		if(diffnsec < 0){
			diffsec--;
			diffnsec += 1000000000;
		}
		printf("%s:%ds %dns\n",str,diffsec,diffnsec);
	}
	return (struct timespec) {diffsec, diffnsec};
}
 

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}
            
// Host code
int main()
{
	// declare timer
	struct timespec timer;

    int N = 1000000000;
    size_t size = N * sizeof(float);

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);


	check_timer(NULL,&timer);
    // Initialize input vectors
    for(int i = 0; i < N; i++){
    	h_A[i] = i;
    	h_B[i] = N - i;
    }
	check_timer("Time to initialize",&timer);

    // Allocate vectors in device memory
    float* d_A;
    hipMalloc(&d_A, size);
    float* d_B;
    hipMalloc(&d_B, size);
    float* d_C;
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
	check_timer(NULL,&timer);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	check_timer("Time to copy to device",&timer);

    // Invoke kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =
            (N + threadsPerBlock - 1) / threadsPerBlock;
	check_timer(NULL,&timer);
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	check_timer("Time to execute kernel",&timer);

    // Copy result from device memory to host memory
    // h_C contains the result in host memory
	check_timer(NULL,&timer);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
	check_timer("Time to copy back to host",&timer);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    int errors = 0;
    for(int i = 0; i < N; i++){
    	if(h_C[i] != N){
    		errors ++;
    	}

    }
    printf("checking done, errors = %d\n");

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);
}